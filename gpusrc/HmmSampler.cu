#include "hip/hip_runtime.h"

#include "HmmSampler.h"
#include <stdio.h>
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>
#include <thrust/scatter.h>
#include <thrust/random/normal_distribution.h>
#include <tuple>
#include <ctime>
#include <utility>
#include <cmath>
#include <algorithm>
#include "cusp/array1d.h"
#include "cusp/array2d.h"
#include "cusp/csr_matrix.h"
#include "cusp/blas/blas.h"
#include "cusp/multiply.h"
#include "cusp/elementwise.h"
#include "cusp/functional.h"
#include "cusp/print.h"
#include "obs_models.h"
#include "myarrays.cu"
#include "Indexer.cu"
#include "State.cu"
#include <chrono> // for testing
#include <random>
#include <limits>
#include <iostream>
#include <iomanip>
#include "cuda_helpers.h"
using namespace cusp;
using namespace std;
typedef std::chrono::high_resolution_clock Clock;
float nano_to_sec = 1.0e-9f;

// typedef std::numeric_limits<float> float_limit;
typedef cusp::array1d<float,cusp::device_memory> cusparray;
typedef cusparray::view ArrayView;

Model::Model(int pi_num_rows, int pi_num_cols, float* pi_vals, int pi_vals_size, int* pi_row_offsets,
int pi_row_offsets_size, int* pi_col_indices, int pi_col_indices_size, float* lex_vals, int lex_vals_size,
int lex_num_rows, int lex_num_cols, int a_max, int b_max, int g_max, int depth, float* pos_vals, int pos_vals_size,
int embed_num_words, int embed_num_dims, int embed_vals_size, float* embed_vals, int EOS_index):
pi_num_rows(pi_num_rows), pi_num_cols(pi_num_cols), pi_vals(pi_vals),
pi_vals_size(pi_vals_size), pi_row_offsets(pi_row_offsets), pi_row_offsets_size(pi_row_offsets_size),
pi_col_indices(pi_col_indices), pi_col_indices_size(pi_col_indices_size), lex_vals(lex_vals),
lex_vals_size(lex_vals_size), lex_num_rows(lex_num_rows), lex_num_cols(lex_num_cols), a_max(a_max),
b_max(b_max), g_max(g_max), depth(depth), pos_vals(pos_vals), pos_vals_size(pos_vals_size),
embed_vals(embed_vals), embed_num_words(embed_num_words), embed_num_dims(embed_num_dims), embed_vals_size(embed_vals_size), EOS_index(EOS_index){
    // cout << "Entering model constructor" << endl;
    pi = new SparseView(pi_num_rows, pi_num_cols, pi_vals_size, pi_row_offsets, pi_col_indices, pi_vals, pi_row_offsets_size, pi_col_indices_size);
    lex = new DenseView(lex_num_rows, lex_num_cols, lex_vals_size, lex_vals);
    embed = new DenseView(embed_num_words, embed_num_dims, embed_vals_size, embed_vals);
    pos = new Array(pos_vals, pos_vals_size);
    // cout << "Done with model constructor" << endl;
}

Model::~Model(){
//     cout << "Entering model destructor" << endl;
    if(lex != NULL){ delete lex;} else{ cout << "lex was already null" << endl;}
    if(pi != NULL) { delete pi;} else{ cout << "pi was already null" << endl;}
    if(pos != NULL) { delete pos;} else{ cout << "pos was already null" << endl;}
    lex = NULL;
    pi = NULL;
    pos = NULL;
    if(embed != NULL) delete embed;
//     cout << "Done with model destructor" << endl;
}

int Model::get_depth(){
    return depth;
}

PosDependentObservationModel::~PosDependentObservationModel(){
//     cout << "PosDep destructor called" << endl;
//    delete p_indexer;
    delete lexMultiplier;
//    cout << "PosDep destructor called" << endl;
}

Sparse * tile(int g_len, int state_size);
void PosDependentObservationModel::set_models(Model * models){
    //cout << "PosDependentObsModel::set_models called" << endl;
    delete p_indexer;
    p_indexer = new Indexer(models);
    delete lexMultiplier;
    lexMultiplier = tile(models->g_max, p_indexer->get_state_size());
    g_size = models->g_max;
    //cout << "PosDependentObsModel::set_models done" << endl;
}

void PosDependentObservationModel::get_probability_vector(int token, Array* retVal){
    //cout << "PosDependentObsModel::get_prob_vec called" << endl;
    Array posProbs(g_size, 0);
    get_pos_probability_vector(token, &posProbs);
    //cout << "pos probs size = " << posProbs.size() << endl;

    // debugging info:
    //cout << "lexMultiplier size = " << lexView.size() << endl;

    multiply(*lexMultiplier, posProbs, *retVal);
    //cout << "PosDependentObsModel::get_prob_vec done with size: " << view.size() << endl;
}

CategoricalObservationModel::~CategoricalObservationModel(){
//     std::cout << "CatObs destructor called" << std::endl;
//    PosDependentObservationModel::~PosDependentObservationModel();
//    delete lexMatrix;
//    std::cout << "CatObs destructor called" << std::endl;
}

void CategoricalObservationModel::set_models(Model * models){
    //cout << "CatObsModel::set_models called" << endl;
    PosDependentObservationModel::set_models(models);
    lexMatrix = models -> lex;
    //cout << "CatObsModel::set_models done" << endl;
}

void CategoricalObservationModel::get_pos_probability_vector(int token, Array* output){
    //cout << "CatObsModel::get_pos_prob called" << endl;
    array2d_view<ValueArrayView, row_major>* lex_view = lexMatrix -> get_view();
    array2d<float, device_memory>::column_view lex_column = lex_view -> column(token);
    //thrust::fill(output->begin(), output->end(), 0);
    thrust::copy(thrust::device, lex_column.begin(), lex_column.end(), output->begin());
    //cout << "CatObsModel::get_pos_prob returning" << endl;
}

void GaussianObservationModel::set_models(Model * models){
    // cout << "GaussianObsModel::set_models called... pre-computing lex matrix" << endl;
    PosDependentObservationModel::set_models(models);
    int numWords = models->embed_num_words;
    
    DenseView* tempLex = models -> lex;
    // lexMatrix = models -> lex;
    embeddings = models -> embed;
    embed_dims = models -> embed_num_dims;
    array2d_view<ValueArrayView, row_major>* embed_view = embeddings -> get_view();

    cout << "There are " << numWords << " words in the vocab with embedding dimension=" << embed_dims << " and " << g_size << " pos tags." << endl;
    int lexMatrixSize = g_size*numWords;
    float *lexMatrixVals = new float[lexMatrixSize];
    lexMatrix = new DenseView(g_size, numWords, g_size*numWords, &lexMatrixVals[0]);

    thrust::device_vector<double> normalizer(embed_dims);
    thrust::device_vector<float> errors(embed_dims);
    thrust::device_vector<float> stdev_squared(embed_dims);
    thrust::device_vector<double> second_factor(embed_dims);
    thrust::device_vector<double> final_prob(embed_dims);
    thrust::device_vector<float> log_prob(embed_dims);
    thrust::device_vector<float> temp(numWords);

    for(int g = 0; g < g_size; g++){
        cout << "Precomputing distribution for pos tag index " << g << endl;
        array2d<float, device_memory>::row_view pos_row = tempLex->get_view()->row(g);
        thrust::device_vector<float> means(pos_row.begin(), pos_row.begin() + embed_dims );
        thrust::device_vector<float> stdevs(pos_row.begin() + embed_dims, pos_row.end());
        // cout << "Means for pos" << g << ":";
        // debug_print_float_vector(means);
        // cout << endl;
        // cout << "Stdevs for pos" << g << ":";
        // debug_print_float_vector(stdevs);
        // cout << endl;

        // cout << "lex_mat: ";
        for(int word_ind = 0; word_ind < numWords; word_ind++){
            array2d<float, device_memory>::row_view embed_vec = embed_view -> row(word_ind);
            // calculate the normalization term (unary function)
            thrust::transform(stdevs.begin(), stdevs.end(), normalizer.begin(), normal_logpdf_firstfactor());
            // calculate the numerator of the exponentiated factor (binary function):
            // cout << "  Calculating squared error term..." << endl;
            thrust::transform(embed_vec.begin(), embed_vec.end(), means.begin(), errors.begin(), normal_logpdf_squarederror());
            // calculate the denominator of the exponentiated factor (unary):
            // cout << "  Calculating two stdevs squared..." << endl;
            thrust::transform(stdevs.begin(), stdevs.end(), stdev_squared.begin(), normal_logpdf_twostdevsquared());
            // calculate the exponentiated term (binary)
            // cout << "  Calculating second factor ratio" << endl;
            thrust::transform(errors.begin(), errors.end(), stdev_squared.begin(), second_factor.begin(), normal_logpdf_secondfactor());
            // finalize output with simple multiplication:
            // cout << "  Calculating final probability vector" << endl;
            thrust::transform(normalizer.begin(), normalizer.end(), second_factor.begin(), final_prob.begin(), thrust::multiplies<double>());
            // take the log probabiility:
            thrust::transform(final_prob.begin(), final_prob.end(), log_prob.begin(), normal_logpdf_log());
            // take the sum of log probabilities:
            lexMatrix->get_view()->operator()(g,word_ind) = thrust::reduce(log_prob.begin(), log_prob.end());

            // cout << lexMatrix->get_view()->operator()(g,word_ind) << " ";

            if(std::isinf(lexMatrix->get_view()->operator()(g,word_ind))){
                cout << "Word index " << word_ind << " has log prob of -inf" << endl;
                cout << "embeddings: " << endl;
                // debug_print_vector(embed_vec);
                cusp::print(embed_vec);
                cout << endl;
                cout << "normalizer: " << endl;
                debug_print_double_vector(normalizer);
                cout << endl;
                cout << "errors: " << endl;
                debug_print_float_vector(errors);
                cout << endl;
                cout << "stdev squared: " << endl;
                debug_print_float_vector(stdev_squared);
                cout << endl;
                cout << "second factor:" << endl;
                debug_print_double_vector(second_factor);
                cout << endl;
                cout << "final prob: " << endl;
                debug_print_double_vector(final_prob);
                cout << endl;
                cout << "log prob: " << endl;
                debug_print_float_vector(log_prob);
                cout << endl;
            }
        }
        // cout << endl;

        // now normalize for this row -- in log space, substract the maximum (least negative) value, then
        // exponentiate, then normalize
        pos_row = lexMatrix->get_view()->row(g);
        float max_prob = *(thrust::max_element(pos_row.begin(), pos_row.end()));
        float min_prob = *(thrust::min_element(pos_row.begin(), pos_row.end()));
        int min_index = thrust::min_element(pos_row.begin(), pos_row.end()) - pos_row.begin();
        // cout << "Max log prob in row " << g << " is " << max_prob << " and min is " << min_prob << " at index " << min_index << endl;
        // cout << "Prob of 'the' is " << pos_row[1] << endl;
        // subtract max:
        thrust::fill(temp.begin(), temp.end(), max_prob);
        thrust::transform(pos_row.begin(), pos_row.end(), temp.begin(), pos_row.begin(), thrust::minus<float>());
        max_prob = *(thrust::max_element(pos_row.begin(), pos_row.end()));
        min_prob = *(thrust::min_element(pos_row.begin(), pos_row.end()));
        // cout << "After max subtract, max log prob in row " << g << " is " << max_prob << " and min is " << min_prob << endl;
        // cout << "Prob of 'the' is " << pos_row[1] << endl;
        // exponentiate:
        thrust::transform(pos_row.begin(), pos_row.end(), pos_row.begin(), cuda_exp());
        // add a small constant so nothing is zero:
        // thrust::fill(temp.begin(), temp.end(), 0.00000000001);
        // thrust::transform(pos_row.begin(), pos_row.end(), temp.begin(), pos_row.begin(), thrust::plus<float>());
        max_prob = *(thrust::max_element(pos_row.begin(), pos_row.end()));
        min_prob = *(thrust::min_element(pos_row.begin(), pos_row.end()));
        // cout << "After exponentiation, max log prob in row " << g << " is " << max_prob << " and min is " << min_prob << endl;
        // cout << "Prob of 'the' is " << pos_row[1] << endl;
        // get normalizer:
        float normalizer = thrust::reduce(pos_row.begin(), pos_row.end());
        thrust::fill(temp.begin(), temp.end(), normalizer);
        // divide:
        thrust::transform(pos_row.begin(), pos_row.end(), temp.begin(), pos_row.begin(), thrust::divides<float>());
        max_prob = *(thrust::max_element(pos_row.begin(), pos_row.end()));
        min_prob = *(thrust::min_element(pos_row.begin(), pos_row.end()));
        // cout << "After normalization, max prob in row " << g << " is " << max_prob << " and min is " << min_prob << endl;
        // cout << "Prob of 'the' is " << pos_row[1] << endl;
        // if(max_prob > 1.0){
        //     cout << "ERROR: Max probability is too high!" << endl;
        // }
        // cout << "Returning probability distribution: " << endl;
        // cusp::print(pos_row);
        // cout << "Setting value " << lexMatrix->get_view()->operator()(g,1) << " when g=" << g << " and word_ind=1" << endl;

    }
    // cout << "After pre-computing, lex matrix contains:" << endl;
    // cusp::print(*lexMatrix->get_view());
    // cout << "GaussianObsModel::set_models done: embedding matrix has dimensionality " << embed_dims << endl;
}

void GaussianObservationModel::get_pos_probability_vector(int token, Array * output){
    array2d_view<ValueArrayView, row_major>* lex_view = lexMatrix -> get_view();
    array2d<float, device_memory>::column_view lex_column = lex_view -> column(token);
    //thrust::fill(output->begin(), output->end(), 0);
    thrust::copy(thrust::device, lex_column.begin(), lex_column.end(), output->begin());
    // cout << "Filled outupt vector with:" << endl;
    // cusp::print(*output);
}
//     // cout << "GaussianObservationModel::get_pos_probability_vector called" << endl;
//     array2d_view<ValueArrayView, row_major>* embed_view = embeddings -> get_view();
//     array2d<float, device_memory>::row_view embed_vec = embed_view -> row(token);
//     int a_max, b_max, g_max;
//     std::tie(a_max, b_max, g_max) = p_indexer -> getVariableMaxes();
//     thrust::fill(output->begin(), output->end(), 0.0);
//
//     (*output)[0] = -std::numeric_limits<float>::max();
//
//     // cout << "  Initializing intermediate vectors" << endl;
//     thrust::device_vector<float> normalizer(embed_dims);
//     thrust::device_vector<float> errors(embed_dims);
//     thrust::device_vector<float> stdev_squared(embed_dims);
//     thrust::device_vector<float> second_factor(embed_dims);
//     thrust::device_vector<float> final_prob(embed_dims);
//     thrust::device_vector<float> log_prob(embed_dims);
//
//     for(int g = 1; g < g_max; g++){
//         // cout << "  Getting prob estimate p(token_" << token << "|POS_" << g << ")" << endl;
//         // cout << "  Loading means" << endl;
//         thrust::device_vector<float> means(lexMatrix->get_view() -> row(g).begin(), lexMatrix->get_view() -> row(g).begin() + embed_dims );
//         // cout << "Means for pos" << g << ":";
//         // debug_print_vector(means);
//         // cout << endl;
//
//         // cout << "  Loading standard deviations" << endl;
//         thrust::device_vector<float> stdevs(lexMatrix->get_view() -> row(g).begin() + embed_dims, lexMatrix->get_view() -> row(g).end());
//         // cout << "Stdevs for pos" << g << ":";
//         // debug_print_vector(stdevs);
//         // cout << endl;
//
//         // cout << "  Calculating normalizing term first..." << endl;
//         // calculate the normalization term (unary function)
//         thrust::transform(stdevs.begin(), stdevs.end(), normalizer.begin(), normal_logpdf_firstfactor());
//         // calculate the numerator of the exponentiated factor (binary function):
//         // cout << "  Calculating squared error term..." << endl;
//         thrust::transform(embed_vec.begin(), embed_vec.end(), means.begin(), errors.begin(), normal_logpdf_squarederror());
//         // calculate the denominator of the exponentiated factor (unary):
//         // cout << "  Calculating two stdevs squared..." << endl;
//         thrust::transform(stdevs.begin(), stdevs.end(), stdev_squared.begin(), normal_logpdf_twostdevsquared());
//         // calculate the exponentiated term (binary)
//         // cout << "  Calculating second factor ratio" << endl;
//         thrust::transform(errors.begin(), errors.end(), stdev_squared.begin(), second_factor.begin(), normal_logpdf_secondfactor());
//         // finalize output with simple multiplication:
//         // cout << "  Calculating final probability vector" << endl;
//         thrust::transform(normalizer.begin(), normalizer.end(), second_factor.begin(), final_prob.begin(), thrust::multiplies<float>());
//         // take the log probabiility:
//         thrust::transform(final_prob.begin(), final_prob.end(), log_prob.begin(), normal_logpdf_log());
//
//         // cout << "Output of distribution for each dimension: " << endl;
//         // debug_print_vector(final_prob);
//
//         // cout << "  Reducing probability vectors across dimensions to POS tag probability." << endl;
//         (*output)[g] = thrust::reduce(log_prob.begin(), log_prob.end());
//         // (*output)[g] = thrust::reduce(final_prob.begin(), final_prob.end(), thrust::multiplies<float>());
//         // cout << "Reduced output for tag: " << (*output)[g] << endl;
//         // (*output)[g] = exp((*output)[g]);
//         // cout << "Exponentiated output: " << exp((*output)[g]) << endl;
//     }
//     // cout << "GaussianObservationModel::get_pos_probability_vector returning:" << endl;
//     // cusp::print(*output);
// }

// taken away the new tensor
template <class AView>
int HmmSampler::get_sample(AView &v){
    float dart;
//    cusp::print(v);
//     cout << "get_sample()" << endl;
    // array1d<float, device_memory> sum_dict(v.size()); // building a new array, maybe not needed
    // this is the equivalent of np.cumsum() or partial_sum in the stl:
    thrust::inclusive_scan(thrust::device, v.begin(), v.end(), sum_dict->begin());
    // cout << "v[0] = " << v[0] << "v[-1] = " << v[v.size()-1] << " with length " << v.size() << endl;
//     cout << "sum_dict[0] = " << (*sum_dict)[0] << "sum_dict[-1] = " << (*sum_dict)[sum_dict->size()-1] << " with length: " << sum_dict->size() <<  endl;
    //cusp::print(*sum_dict);
//     cout << "sum done" << endl;
    // dart =  static_cast <float> (rand()) / static_cast <float> (RAND_MAX);// / RAND_MAX;
    int dart_target;
    int condition = 1;
    while (condition) {
        dart = dist(mt);
        if (dart != 0.0f and dart != 1.0f) {
//            cout << "dart: "<< scientific << dart << endl;
           dart_target = thrust::upper_bound(thrust::device, sum_dict->begin(), sum_dict->end(), dart) - sum_dict->begin();
           if (dart_target == 0) {
           print(*sum_dict);
           }
//           cout << "dart target (summed): " << dart_target << " " << scientific << (*sum_dict)[dart_target - 1] << endl; //<< " " << scientific << (*sum_dict)[dart_target] << " " << scientific  << (*sum_dict)[dart_target+ 1] << endl;
//           cout << "dart target (summed): " << dart_target << " " << scientific << v[dart_target - 1] << " " << scientific << v[dart_target] << " " << scientific  << v[dart_target+ 1] << endl;
//            cout << "dart target (summed): " << dart_target << " with v size=" << v.size() << endl;
//            float minus_one = (*sum_dict)[dart_target - 1];
//            printf( "%A" , minus_one);
//            cout  << " ";
//            float on_target = (*sum_dict)[dart_target];
//            printf("%A", on_target);
//            cout << " ";
//            float plus_one = (*sum_dict)[dart_target+ 1];
//            printf("%A\n", plus_one);
           if (dart_target != v.size() && v[dart_target] != 0.0f ){
                condition = 0;
//                cout << "out of loop" << endl;
           }
        }
     }
//      cout << "get_sample() done." << endl;
//      cout << "dart_target: " << dart_target << endl;
     return dart_target;
}



// template <class AView>
// void exp_array(AView & v){
//    cuda_exp x;
//    thrust::transform(v.begin(), v.end(), v.begin(), x);
// }

// this gives a TRANSPOSED version of a tile matrix for computation convenience
Sparse * tile(int g_len, int state_size){

//     cout << "5.1" << endl;
//    int copy_times = state_size / g_len;
    // cout << "5.2" << endl;
    int nnz = state_size;
//     cout << "5.3" << endl;
    // cout << g_len << " " << state_size << " " << nnz << endl;
    Sparse * temp_mat = new Sparse(state_size, g_len, nnz); //TRANSPOSED
    blas::fill(temp_mat->values, 1.0f);
//     cout << "5.4" << endl;
    calc_sparse_column temp_func;
    thrust::tabulate(thrust::device, temp_mat->column_indices.begin(), temp_mat->column_indices.end(), temp_func);
    thrust::sequence(thrust::device, temp_mat->row_offsets.begin(), temp_mat->row_offsets.end());
    // for (int i = 0; i <= state_size; i++){
    //     if (i < state_size){
    //         temp_mat->row_offsets[i] = i;
    //     } else {
    //         temp_mat->row_offsets[i] = nnz;
    //     }
    // }
//     cout << "5.5" << endl;

    return temp_mat;
}

// this gives a TRANSPOSED version of matrix repetition for computation convenience
Sparse * expand(int g_len, int state_size){

//     cout << "expand5.1" << endl;
    int col_size = state_size / g_len;
//     cout << "expand5.2" << endl;
    int nnz = state_size;
//     cout << "expand5.3" << endl;
    // cout << g_len << " " << state_size << " " << nnz << endl;
    Sparse * temp_mat = new Sparse(state_size, col_size, nnz); //TRANSPOSED
    blas::fill(temp_mat->values, 1.0f);
//     cout << "expand5.4" << endl;
    g_integer_division temp_func;
    thrust::tabulate(thrust::device, temp_mat->column_indices.begin(), temp_mat->column_indices.end(), temp_func);
    thrust::sequence(thrust::device, temp_mat->row_offsets.begin(), temp_mat->row_offsets.end());
//     cout << "expand5.5" << endl;
    return temp_mat;
}

void get_row(csr_matrix_view<IndexArrayView, IndexArrayView, ValueArrayView>* s, int i, Array& result,
Array* pos_full_array, int g_max, int b_max){
    blas::fill(result, 0.0f);
    int pi_row_index = i / g_max;
//    cout <<"get row in"<< endl;
//    cout << pi_row_index << "pi row index" << endl;
    if (s->row_offsets[pi_row_index] - s->row_offsets[pi_row_index+1] != 0){
        int num_of_entry = s->row_offsets[pi_row_index+1] - s->row_offsets[pi_row_index];
        int start_column_index = s->row_offsets[pi_row_index];
        IndexArrayView column_indices = s -> column_indices.subarray(start_column_index, num_of_entry);
        ValueArrayView values = s -> values.subarray(start_column_index, num_of_entry);
        thrust::scatter(thrust::device, values.begin(), values.end(), column_indices.begin(), result.begin());
//        cusp::print(result);
//        cout << "g is " << g_index << "and the prob is " <<(*pos_matrix)[pos_matrix_dim] << endl;
//        blas::scal(result, (*pos_full_array)[i]);
    }
//    cout << "get row out" << endl;
}

int get_max_len(std::vector<std::vector<int> > sents){
//    cout <<"get_max_len in"<< endl;
    int max_len = 0;
    for(std::vector<int> sent : sents){
        if(sent.size() > max_len){
            max_len = sent.size();
        }
    }
//    cout << "get_max_len out" << endl;

    return max_len;
}

Dense* get_sentence_array(std::vector<std::vector<int> > sents, int max_len){
    Dense* array = new Dense( sents.size(), max_len, 0 );
//    cout <<"get_sentence_array in"<< endl;
    for(int i = 0; i < sents.size(); i++){
      std::vector<int> sent = sents[i];
      for(int token_ind = 0; token_ind < sent.size(); token_ind++){
        array -> operator()(i, token_ind) = sent[token_ind];
      }
    }
//    cout <<"get_sentence_array out"<< endl;
    return array;

}

void HmmSampler::set_models(Model * models){

//     cout << "set_models 1" << endl;
    p_model = models;
//     cout << "set_models 2" << endl;
    if (p_indexer != NULL){
        delete p_indexer;
	p_indexer = NULL;
    }
    p_indexer = new Indexer(models);
//     cout << "set_models 3" << endl;
    int g_len = p_model-> g_max;
    int state_size = p_indexer -> get_state_size();
    hipMemset(&G_SIZE,0,sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(G_SIZE), &g_len, sizeof(int), 0, hipMemcpyHostToDevice);
//     cout << "set_models 4" << endl;
    //if (lexMatrix != NULL){
    //    delete lexMatrix;
    //    lexMatrix = NULL;
    //}

    // print(*(lexMatrix->get_view()));
    // exp_array(lexMatrix->get_view()->values); // exp the lex dist // the gpu models are not logged, should not need this
//     cout << "set_models 5" << endl;
    pos_matrix = p_model -> pos;
//    cusp::print(*pos_matrix);
//    cout << "set_models 6" << endl;
    expand_mat = expand(g_len, p_indexer->get_state_size());
//    cout << "set_models 7" << endl;
    // print(*lexMultiplier);
    pi = p_model -> pi;
    // print( *(pi->get_view()) );
    if (trans_slice != NULL){
        delete trans_slice;
        trans_slice = NULL;
    }
    trans_slice = new Array(p_indexer->get_state_size(), 0.0f);
//    cout << "set_models 8" << endl;
    //expanded_lex = trans_slice;
    sum_dict = trans_slice;
    int b_len = p_model -> b_max;
//    cout << "pos full array" << endl;
    int depth = p_model -> get_depth();
    pos_full_array = make_pos_full_array(pos_matrix, g_len, b_len, depth, state_size);
    //cout << "set_models 9" << endl;
//    print(*pos_full_array);
    // cout.precision(float_limit::max_digits10);
    obs_model->set_models(models);
}

Array* HmmSampler::make_pos_full_array(Array* pos_matrix ,int g_max, int b_max, int depth, int state_size){
//    cout << "make_pos_full_array in" << endl;
    int pos_matrix_size = pow(b_max, depth) * g_max * 2;
    int copy_times = state_size / pos_matrix_size;
    Array* temp_array = new Array(state_size, 0.0f);
    for (int i = 0; i < copy_times; i ++){
//        cout << "calc full pos " << i  << " from " << i*pos_matrix_size << " to " << (i+1)*pos_matrix_size <<
//        " total " << copy_times << endl;
        ArrayView one_section_of_array(temp_array->subarray(i*pos_matrix_size, pos_matrix_size));
        copy(*pos_matrix, one_section_of_array);
    }
//    cout << "make_pos_full_array out" << endl;
//    cout << " THE VALUE " << (*temp_array)[26426] << endl;
    return temp_array;
}

void HmmSampler::initialize_dynprog(int batch_size, int max_len){
    try{
//    cout << "initialize_dynprog in" << endl;
    sampler_batch_size = batch_size;
    max_sent_len = max_len;
    dyn_prog = new Dense*[max_len];
    for(int i = 0; i < max_len; i++){
        dyn_prog[i] = new Dense(p_indexer->get_state_size(), batch_size, 0.0f);
    }
//    cout << "initialize_dynprog 2" << endl;
    start_state = new Dense(p_indexer->get_state_size(), batch_size, 0.0f);
    for(int i = 0; i < batch_size; i++){
        start_state->operator()(0, i) = 1;
    }
    int a_max, b_max, g_max;
//    cout << "initialize_dynprog 3" << endl;
    std::tie(a_max, b_max, g_max) = p_indexer -> getVariableMaxes();
    int state_size_no_g = p_indexer->get_state_size() / g_max;
    dyn_prog_part = new Dense(state_size_no_g, batch_size, 0.0f);
//    cout << "initialize_dynprog out" << endl;
    } catch (...) {
        cout << "init dynprog error!" << endl;
        throw;
    }
}

void HmmSampler::g_factored_multiply(Dense* prev_dyn_prog_slice, Dense* this_dyn_prog_slice){
//    cout << "factor multiply 1" << endl;
    csr_matrix_view<IndexArrayView,IndexArrayView,ValueArrayView>* curtailed_transition = pi -> get_view();
    multiply(*curtailed_transition, *prev_dyn_prog_slice, *dyn_prog_part);
//    cout << "dyn prog part" << endl;
//    print(*dyn_prog_part);
//    cout << "factor multiply 2" << endl;
    int a_max, b_max, g_max;
    std::tie(a_max, b_max, g_max) = p_indexer -> getVariableMaxes();
    int state_size_no_g = p_indexer->get_state_size() / g_max;
    int state_size = p_indexer->get_state_size();
    multiply(*expand_mat, *dyn_prog_part, *this_dyn_prog_slice);
//    cout << "factor multiply 3" << endl;
//    cout << "this dyn prog slice" << endl;
//    print(*this_dyn_prog_slice);
    for(int i = 0; i < sampler_batch_size; i++ ){ // this is not efficient. maybe there is a better way
//        cout << "insde g_multipy i " << i << endl;
        array2d<float, device_memory>::column_view this_dyn_prog_slice_column = this_dyn_prog_slice->column(i);
//        blas::xmy(this_dyn_prog_slice_column, *pos_matrix, this_dyn_prog_slice_column);
//        print(this_dyn_prog_slice_column);
//        print(*pos_full_array);
        blas::xmy(this_dyn_prog_slice_column, *pos_full_array ,this_dyn_prog_slice_column);
//        cout << "this dyn prog slice column" << endl;
//        print(this_dyn_prog_slice_column);

    }
//    cout << "factor multiply 4" << endl;
}

std::vector<float> HmmSampler::forward_pass(std::vector<std::vector<int> > sents, int sent_index){
    // auto t1 = Clock::now();
    // cout << "Starting forward batch from sentence index: " << sent_index << endl;
    float normalizer;
    int a_max, b_max, g_max; // index, token, g_len;
    std::tie(a_max, b_max, g_max) = p_indexer -> getVariableMaxes();
    std::vector<int> sent = sents[0];
    std::vector<float> log_probs;
    int batch_size = sents.size();
    int batch_max_len = get_max_len(sents);
    // np_sents is |batches| x max_len
    Dense* np_sents = get_sentence_array(sents, batch_max_len);
    Array expanded_lex(p_indexer->get_state_size(), 0);
    Array temp_vec(p_indexer->get_state_size(), 0);

    //array2d_view<ValueArrayView, row_major>* lex_view = lexMatrix -> get_view();
//    cout << "Forward in 2 " << endl;
    // initialize likelihood vector:
    for(int sent_ind = 0; sent_ind < sents.size(); sent_ind++){
        log_probs.push_back(0);
    }


    for(int ind = 0; ind < batch_max_len; ind++){
        // cout << "Processing token index " << ind << " for " << batch_size << " sentences." << endl;
        Dense *cur_mat = dyn_prog[ind];
        Dense *prev_mat;

        if(ind == 0){
            prev_mat = start_state;
        }else{
            // Grab the ind-1th row of dyn_prog and multiply it by the transition matrix and put it in
            // the ind^th row.
            prev_mat = dyn_prog[ind-1];
        }

        // prev_mat is |states| x |batches| at time ind-1
        // pi_view is |states| x |states| transition matrix with time t on rows and t-1 on columns (i.e. transposed)
        // so after this multiply cur_mat is |states| x |batches| incorporating transition probabilities
        // but not evidence
//        cout << "Performing transition multiplication" << endl;
        g_factored_multiply(prev_mat, cur_mat);
//        print(cur_mat[0]);
//        cout << "Done with transition" << endl;
//        cout << "performing observation multiplications" << endl;

        auto trans_done = Clock::now();

        // for now incorporate the evidence sentence-by-sentence:
        for(int sent_ind = 0; sent_ind < sents.size(); sent_ind++){
            // not every sentence in the batch will need the full batch size
            if(sents[sent_ind].size() <= ind){
                continue;
            }
            // cout << "Processing evidence for sentence index " << sent_ind << endl;
            int token = sents[sent_ind][ind];

            // dyn_prog_row is 1 x state_size
            // dyn_prog_column is state_size x 1
            array2d<float, device_memory>::column_view dyn_prog_col = cur_mat->column(sent_ind);
            // cout << "Getting observation probability for token " << token << endl;
            obs_model->get_probability_vector(token, &expanded_lex);
            float max_prob = *(thrust::max_element(expanded_lex.begin(), expanded_lex.end()));
            // cout << "Max value of observation model is " << max_prob << endl;

            if(max_prob < 0){
                // cout << "Max prob is < 0 so I'm exp'ing" << endl;
                thrust::transform(expanded_lex.begin(), expanded_lex.end(), expanded_lex.begin(), cuda_exp());
            }
            // cout << "observation probabilities after expansion and (possible) exponentiation" << endl;
            // cusp::print(expanded_lex);

            thrust::copy(dyn_prog_col.begin(), dyn_prog_col.end(), temp_vec.begin());
            blas::xmy(expanded_lex, dyn_prog_col, dyn_prog_col);

//            cout << "Computing normalizer" << endl;
            normalizer = thrust::reduce(thrust::device, dyn_prog_col.begin(), dyn_prog_col.end());
            // cout << "Normalizing over col with result: " << normalizer << endl;
            if(std::isnan(normalizer)){
              cout << "Found a normalizer that is nan!" << endl;
              cout << "Dyn prog col before multiply: " << endl;
              cusp::print(temp_vec);
              cout << "expanded lex probability vector: " << endl;
              cusp::print(expanded_lex);
              cout << "Dynamic programming matrix column:" << endl;
              cusp::print(dyn_prog_col);
            }
//            cout << "Scaling by normalizer" << endl;
            blas::scal(dyn_prog_col, 1.0f/normalizer);
//            cout << "Adding normalizer" << endl;
            log_probs[sent_ind] += log10f(normalizer);

            if (sents[sent_ind].size() - 1 == ind){
                int EOS = p_indexer -> get_EOS_full();
                // cout << EOS << endl;
                array2d<float, device_memory>::column_view final_dyn_col = cur_mat->column(sent_ind);
//                cout << cusp::blas::asum(final_dyn_col) << endl;
                get_row(pi->get_view(), EOS, *trans_slice, pos_full_array, g_max, b_max);
//                cout << cusp::blas::asum(*trans_slice) << endl;
                float final_normalizer = cusp::blas::dot(*trans_slice, final_dyn_col);
                log_probs[sent_ind] += log10f(final_normalizer);
                // cout << " ind "<< ind << " sent_ind " << sent_ind << " end log prob " << log10f(final_normalizer) << endl;
            }
        }

        auto norm_done = Clock::now();
    }
//    for (int i =0; i < sents[0].size(); i++){
//    print(*dyn_prog[i]);
//    }
//    cout << "end of dyn prog" << endl;
    // cout << "Finished forward pass (cuda) and returning vector with " << log_probs.size() << " elements." << endl;
    return log_probs;
}

std::vector<std::vector<State> > HmmSampler::reverse_sample(std::vector<std::vector<int>> sents, int sent_index){
//     cout << "Reverse sampling batch with starting sent index of " << sent_index << endl;
    // auto t2 = Clock::now();
    std::vector<std::vector<State>> sample_seqs;
    std::vector<State> sample_seq;
    std::vector<int> sample_t_seq;
    int sample_t; // , t, ind; totalK, depth,
    int batch_size = sents.size();
    int batch_max_len = get_max_len(sents);
    // int prev_depth, next_f_depth, next_awa_depth;
    //float sample_log_prob;//trans_prob,
    // float t0, t1;
//    array1d<float, device_memory> trans_slice;
    State sample_state;
    //sample_log_prob = 0.0f;
//    std::vector<int> fake_ts = {25,218,21,213,25,230,42,22,38,229,233,41,25,154,2};
//    reverse(fake_ts.begin(), fake_ts.end());
//    for(std::vector<int> sent : sents){
    for(int sent_ind = 0; sent_ind < batch_size; sent_ind++){
        sample_seq = std::vector<State>();
//        cout << "Processing sentence " << sent_ind << " of the batch" << endl;
        std::vector<int> sent = sents[sent_ind];
//        for(int token_ind = 0; token_ind < sent.size(); token_ind++){
//          cout << sent[token_ind] << " ";
//        }
//        cout << endl;

        // Start with EOS
//        if (sent.size() == 1) {
//            sample_t = p_indexer->get_EOS_1wrd_full();
//        } else {
            sample_t = p_indexer->get_EOS_full();
//        }

        for (int t = sent.size() - 1; t > -1; t --){
//            cout << "t" << t << " prev sample t is " << sample_t <<endl;
            // auto t11 = Clock::now();
            std::tie(sample_state, sample_t) = _reverse_sample_inner(sample_t, t, sent_ind);
//             cout << "Sample t is " << sample_t << endl;
//             cout << sample_state.f << " " << sample_state.j << " " << sample_state.a[0] << " " << sample_state.a[1] << " " << sample_state.b[0] << " " << sample_state.b[1] << " " << sample_state.g << endl;
            if(!sample_state.depth_check()){
              cout << "Depth error in state assigned at index" << t << endl;
            }
            sample_seq.push_back(sample_state);
            sample_t_seq.push_back(sample_t);
            // auto t12 = Clock::now();
//             cout << "backpass2inside: " << (float)std::chrono::duration_cast<std::chrono::nanoseconds>(t12 - t11).count() * nano_to_sec << " s" << endl;

        }
        // auto t4 = Clock::now();
        std::reverse(sample_seq.begin(), sample_seq.end());
//        cout << "finished backward sampling of a sentence" << endl;
        //cout << sample_seq->size() << endl;
        // auto t5 = Clock::now();
        // cout << "backpass1: " << (float)std::chrono::duration_cast<std::chrono::nanoseconds>(t3 - t2).count() * nano_to_sec << " s" << endl;
        // cout << "backpass2: " << (float)std::chrono::duration_cast<std::chrono::nanoseconds>(t4 - t3).count() * nano_to_sec << " s" << endl;
        // cout << "backpass: " << (float)std::chrono::duration_cast<std::chrono::nanoseconds>(t5 - t2).count() * nano_to_sec << " s" << endl;
        //for (int k : sample_t_seq){
        //    cout << sent_index << " : " << k  << endl;
        //}
        sample_seqs.push_back(sample_seq);
    }
//    for(auto i : sample_t_seq){
//        cout << i << endl;
//    }
//    for (int i =0; i < sents[0].size(); i++){
//    print(*dyn_prog[i]);
//    }
//    cout << "end of dyn prog" << endl;
//    cout << "Done with reverse()" << endl;
    return sample_seqs;
}


std::tuple<State, int> HmmSampler::_reverse_sample_inner(int& sample_t, int& t, int sent_ind){
    //int ind;
    float normalizer;
    // auto t11 = Clock::now();
//    cout << "before get row" << endl;
//    int prev_sample_t = sample_t;
    int g_max =  p_model -> g_max;
    int b_max = p_model -> b_max;
    get_row(pi->get_view(), sample_t, *trans_slice, pos_full_array, g_max, b_max);
    // print(*trans_slice);
    // auto t12 = Clock::now();
    array2d<float, device_memory>::column_view dyn_prog_col = dyn_prog[t]->column(sent_ind);
    float dyn_prog_col_sum = thrust::reduce(thrust::device, dyn_prog_col.begin(), dyn_prog_col.end());
//     cout << "Dyn_prog_row " << dyn_prog_col_sum << endl;
    // print(dyn_prog_row);
    float trans_slice_sum = thrust::reduce(thrust::device, (*trans_slice).begin(), (*trans_slice).end());
//    cout << "trans_slice " << trans_slice_sum << endl;
    //if (trans_slice_sum != 0.0f){
        cusp::blas::xmy(*trans_slice, dyn_prog_col, dyn_prog_col);
    //}
    // auto t13 = Clock::now();
//    cusp::array1d<float, host_memory> un_normalized_sums(dyn_prog_col);
    normalizer = thrust::reduce(thrust::device, dyn_prog_col.begin(), dyn_prog_col.end());
//     cout << "normalizer " << normalizer <<endl;
    blas::scal(dyn_prog_col, 1.0f/normalizer);
    // thrust::transform(dyn_prog_row.begin(), dyn_prog_row.end(), dyn_prog_row.begin(), multiplies_value<float>(1 / normalizer));
    // auto t14 = Clock::now();
    sample_t = get_sample(dyn_prog_col);
//    cout << "sample_t "<< sample_t << endl;
//    sample_t = fake_ts.back();
//    fake_ts.pop_back();
    //if (sample_t == 0){
    //    print(dyn_prog_row);
    //}
    // auto t15 = Clock::now();
    State sample_state = p_indexer -> extractState(sample_t);
//    cout << "done with reverse sampler inner" << endl;
    // if ((sample_state.a[1] == 0 && sample_state.b[1] != 0)|| (sample_state.a[1] != 0 && sample_state.b[1] == 0) ){
//    get_row(pi->get_view(), prev_sample_t, *trans_slice); // why is this used?
        // cout << "Dyn Prog(normalized): " << scientific << dyn_prog_row[sample_t -1] << " " << scientific << dyn_prog_row[sample_t] <<" "<< scientific <<dyn_prog_row[sample_t + 1] <<endl;
        // cout << "Trans Slice (Pi Row): " <<" " << scientific << (*trans_slice)[sample_t-1]<< " "<< scientific << (*trans_slice)[sample_t] <<" " << scientific << (*trans_slice)[sample_t+1]<< endl;
        // cout << "Prefix Sum(unnormalized, before sum): " << scientific << un_normalized_sums[sample_t-1] << " " << scientific << un_normalized_sums[sample_t] << " " << scientific << un_normalized_sums[sample_t + 1] << endl;
        // cout << "Prefix Sum(unnormalozed, before sum): ";
        // float temp_1 = un_normalized_sums[sample_t-1];
        // printf( "%A", temp_1);
        // cout << " ";
        // float temp_2 = un_normalized_sums[sample_t];
        // printf("%A", temp_2);
        // cout << " ";
        // float temp_3 = un_normalized_sums[sample_t+1];
        // printf("%A\n", temp_3);
        // cout << "sample_t: " << sample_t << endl;

    // }
    // cout << "backpass1reverseinner: " << (float)std::chrono::duration_cast<std::chrono::nanoseconds>(t12 - t11).count() * nano_to_sec << " s" << endl;
    // cout << "backpass1reverseinner: " << (float)std::chrono::duration_cast<std::chrono::nanoseconds>(t13 - t12).count() * nano_to_sec << " s" << endl;
    // cout << "backpass1reverseinner: " << (float)std::chrono::duration_cast<std::chrono::nanoseconds>(t14 - t13).count() * nano_to_sec << " s" << endl;
    // cout << "backpass1reverseinner: " << (float)std::chrono::duration_cast<std::chrono::nanoseconds>(t15 - t14).count() * nano_to_sec << " s" << endl;

    return std::make_tuple(sample_state, sample_t);
}

std::tuple<std::vector<std::vector<State> >, std::vector<float>> HmmSampler::sample(std::vector<std::vector<int>> sents, int sent_index) {
    std::vector<float> log_probs;
    std::vector<std::vector<State> > states;

    try{
        log_probs = forward_pass(sents, sent_index);
    }catch(thrust::system_error &e){
        cerr << "Error in forward pass: " << e.what() << endl;
        throw e;
    }
    try{
        states = reverse_sample(sents, sent_index);
    }catch(thrust::system_error &e){
        cerr << "Error in reverse sample: " << e.what() << endl;
        throw e;
    }


    return std::make_tuple(states, log_probs);
}


HmmSampler::HmmSampler() : HmmSampler(std::time(0), ModelType::CATEGORICAL_MODEL){
    //cout << "Empty constructor called, initializing with defaults: seed=0 and categorical model" << endl;
}
HmmSampler::HmmSampler(int seed) : HmmSampler(seed, ModelType::CATEGORICAL_MODEL) {
    //cout << "One-arg constructor called, initializing with default categorical model" << endl;
}
HmmSampler::HmmSampler(int seed, ModelType model_type) : seed(seed){
    //cout << "Two-arg constructor called for sampler" << samplerNum << endl;
    if (seed == 0){
        mt.seed(std::time(0));
    } else{
        mt.seed(seed);
    }

    if (model_type == ModelType::CATEGORICAL_MODEL) {
        // cout << "Creating categorical model" << endl;
        obs_model = new CategoricalObservationModel();
    }else if(model_type == ModelType::GAUSSIAN_MODEL) {
        cout << "Creating gaussian model" << endl;
        obs_model = new GaussianObservationModel();
    }
}

HmmSampler::~HmmSampler(){
//    cout << "HmmSampler destructor called for sampler "  << endl;
    if(dyn_prog != NULL){
        for(int i = 0; i < max_sent_len; i++){
            delete dyn_prog[i];
        }
        delete[] dyn_prog;
    }
    delete start_state;
    //delete[] dyn_prog;
//    delete p_model;
    delete p_indexer;
    //delete lexMatrix;
    //delete dyn_prog;
    //delete lexMultiplier;
    //delete pi;
    delete trans_slice;
    //delete expanded_lex;
    //delete sum_dict;
    delete expand_mat;
    delete dyn_prog_part;
    delete pos_full_array;
    if(obs_model != NULL) {delete obs_model;}
}
