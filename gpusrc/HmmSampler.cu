#include "hip/hip_runtime.h"

#include "HmmSampler.h"
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>
#include <thrust/scatter.h>
#include <tuple>
#include <ctime>
#include <utility>
#include <cmath>
#include <algorithm>
#include "cusp/array1d.h"
#include "cusp/array2d.h"
#include "cusp/csr_matrix.h"
#include "cusp/blas/blas.h"
#include "cusp/multiply.h"
#include "cusp/elementwise.h"
#include "cusp/functional.h"
#include "cusp/print.h"
#include "myarrays.cu"
#include "Indexer.cu"
#include "State.cu"
#include <chrono> // for testing
#include <random>

using namespace cusp;
using namespace std;
typedef std::chrono::high_resolution_clock Clock;
float nano_to_sec = 1.0e-9f;
__device__ int STATE_SIZE;

Model::Model(int pi_num_rows, int pi_num_cols, float* pi_vals, int pi_vals_size, int* pi_row_offsets, int pi_row_offsets_size, int* pi_col_indices, int pi_col_indices_size, float* lex_vals, int lex_vals_size, int lex_num_rows, int lex_num_cols, int a_max, int b_max, int g_max, int depth): pi_num_rows(pi_num_rows), pi_num_cols(pi_num_cols), pi_vals(pi_vals), pi_vals_size(pi_vals_size), pi_row_offsets(pi_row_offsets), pi_row_offsets_size(pi_row_offsets_size), pi_col_indices(pi_col_indices), pi_col_indices_size(pi_col_indices_size), lex_vals(lex_vals), lex_vals_size(lex_vals_size), lex_num_rows(lex_num_rows), lex_num_cols(lex_num_cols), a_max(a_max), b_max(b_max), g_max(g_max), depth(depth){
    pi = new SparseView(pi_num_rows, pi_num_cols, pi_vals_size, pi_row_offsets, pi_col_indices, pi_vals, pi_row_offsets_size, pi_col_indices_size);
    lex = new DenseView(lex_num_rows, lex_num_cols, lex_vals_size, lex_vals);
}

Model::~Model(){
    delete lex;
    delete pi;
}

int Model::get_depth(){
    return depth;
}

// taken away the new tensor
template <class AView>
int HmmSampler::get_sample(AView &v){
    float dart;
    // array1d<float, device_memory> sum_dict(v.size()); // building a new array, maybe not needed
    thrust::inclusive_scan(thrust::device, v.begin(), v.end(), sum_dict->begin());
    // dart =  static_cast <float> (rand()) / static_cast <float> (RAND_MAX);// / RAND_MAX;
    dart = dist(mt);
    // cout << "dart"<< dart << endl;
    return thrust::lower_bound(thrust::device, sum_dict->begin(), sum_dict->end(), dart)  - sum_dict->begin();
}

class cuda_exp : public thrust::unary_function<float, float>{
public:
   __host__ __device__
   float operator()(float x){
       return pow(10,x);
   }
};
class calc_sparse_column : public thrust::unary_function<int, int>{
public:
   __host__ __device__
   int operator()(int x){
       return x % STATE_SIZE;
   }
};
template <class AView>
void exp_array(AView & v){
   cuda_exp x;
   thrust::transform(v.begin(), v.end(), v.begin(), x);
}

// this gives a TRANSPOSED version of a tile matrix for computation convenience
Sparse * tile(int g_len, int state_size){

    // cout << "5.1" << endl;
    int copy_times = state_size / g_len;
    // cout << "5.2" << endl;
    int nnz = g_len * copy_times;
    // cout << "5.3" << endl;
    // cout << g_len << " " << state_size << " " << nnz << endl;
    Sparse * temp_mat = new Sparse(state_size, g_len, nnz); //TRANSPOSED
    blas::fill(temp_mat->values, 1.0f);
    // cout << "5.4" << endl;
    calc_sparse_column temp_func;
    thrust::tabulate(thrust::device, temp_mat->column_indices.begin(), temp_mat->column_indices.end(), temp_func);
    thrust::sequence(thrust::device, temp_mat->row_offsets.begin(), temp_mat->row_offsets.end());
    // for (int i = 0; i <= state_size; i++){
    //     if (i < state_size){
    //         temp_mat->row_offsets[i] = i;
    //     } else {
    //         temp_mat->row_offsets[i] = nnz;
    //     }
    // }
    // cout << "5.5" << endl;
    return temp_mat;
}

// build a dense array from a row in a sparse matrix
// void get_row(csr_matrix_view<IndexArrayView, IndexArrayView, ValueArrayView>* s, int i, Array& result){
//     blas::fill(result, 0.0f);
//     if (s->row_offsets[i] - s->row_offsets[i+1] != 0){
//         int num_of_entry = s->row_offsets[i+1] - s->row_offsets[i];
//         int start_column_index = s->row_offsets[i];
//         for (i = 0; i < num_of_entry; i ++){
//             result[s->column_indices[start_column_index + i]] = s->values[start_column_index + i];
//         }
//     }
// }
// using scatter
void get_row(csr_matrix_view<IndexArrayView, IndexArrayView, ValueArrayView>* s, int i, Array& result){
    blas::fill(result, 0.0f);
    if (s->row_offsets[i] - s->row_offsets[i+1] != 0){
        int num_of_entry = s->row_offsets[i+1] - s->row_offsets[i];
        int start_column_index = s->row_offsets[i];
        IndexArrayView column_indices = s -> column_indices.subarray(start_column_index, num_of_entry);
        ValueArrayView values = s -> values.subarray(start_column_index, num_of_entry);
        thrust::scatter(thrust::device, values.begin(), values.end(), column_indices.begin(), result.begin());
    }
}

void HmmSampler::set_models(Model * models){
    // cout << '1' << endl;
    p_model = models;
    // cout << '2' << endl;
    if (p_indexer != NULL){
        delete p_indexer;
	p_indexer = NULL;
    }
    p_indexer = new Indexer(models);
    // cout << '3' << endl;
    int g_len = p_model-> g_max;
    hipMemset(&STATE_SIZE,0,sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(STATE_SIZE), &g_len, sizeof(int), 0, hipMemcpyHostToDevice);
    // cout << '4' << endl;
    //if (lexMatrix != NULL){
    //    delete lexMatrix;
    //    lexMatrix = NULL;
    //}
    lexMatrix = p_model -> lex; 
    // print(*(lexMatrix->get_view()));
    // exp_array(lexMatrix->get_view()->values); // exp the lex dist // the gpu models are not logged, should not need this
    // cout << '5' << endl;
    if (lexMultiplier != NULL){
        delete lexMultiplier;
        lexMultiplier = NULL;
    }
    lexMultiplier = tile(g_len, p_indexer->get_state_size());
    // print(*lexMultiplier);
    pi = p_model -> pi;
    // print( *(pi->get_view()) );
    if (trans_slice != NULL){
        delete trans_slice;
        trans_slice = NULL;
    }
    trans_slice = new Array(p_indexer->get_state_size(), 0.0f);
    expanded_lex = trans_slice;
    sum_dict = trans_slice;
}

void HmmSampler::initialize_dynprog(int max_len){
    // cout << '2' << endl;
    if (dyn_prog != NULL){
        delete dyn_prog;
    }
    dyn_prog = new Dense( max_len, p_indexer->get_state_size(), 0.0f );
}

float HmmSampler::forward_pass(std::vector<int> sent, int sent_index){
    // auto t1 = Clock::now();
    float sentence_log_prob, normalizer;
    int a_max, b_max, g_max; // index, token, g_len;
    std::tie(a_max, b_max, g_max) = p_indexer -> getVariableMaxes();
    
    // no need to matrixize the dyn_prog
    // assume sent is a vector of word indices
    int i = 0;
    // array1d<float, device_memory> temp_dyn_prog_row(p_indexer -> get_state_size());
    blas::fill(dyn_prog -> values, 0.0f);
    csr_matrix_view<IndexArrayView,IndexArrayView,ValueArrayView>* pi_view = pi -> get_view();
    array2d_view<ValueArrayView, row_major>* lex_view = lexMatrix -> get_view();
    for (int token : sent){
        // cout << i << endl;
        array2d<float, device_memory>::row_view dyn_prog_row = dyn_prog->row(i);
        if (i == 0){
            array2d<float, device_memory>::column_view lex_column = (lexMatrix->get_view())->column(token);
            // print(lex_column);
            array2d<float, device_memory>::column_view no_head_tail_column = lex_column.subarray(1, lex_column.size() - 2);
            array2d<float, device_memory>::row_view dyn_prog_row_section = dyn_prog_row.subarray(1, g_max - 2);
            copy(no_head_tail_column, dyn_prog_row_section); // memory copy of array views
            // print(dyn_prog_row_section);
            // print(no_head_tail_column);
            // print(dyn_prog_row_section);
            // print(dyn_prog_row);
        } else {
            // get a slice of forward matrix
            // transition
            array2d<float, device_memory>::row_view dyn_prog_prev_row = dyn_prog->row(i - 1);
            // print(dyn_prog_prev_row);
            // transpose PI
            // csr_matrix_view<IndexArrayView, IndexArrayView, ValueArrayView>& pi_address = *(pi->get_view());
            // array2d_view<ValueArrayView, row_major> dyn_prog_prev_row_2dview = make_array2d_view(1, dyn_prog_prev_row.size(), dyn_prog_prev_row.size(), dyn_prog_prev_row, row_major());
            // array2d_view<ValueArrayView, row_major> dyn_prog_row_2dview = make_array2d_view(1, dyn_prog_row.size(), dyn_prog_row.size(), dyn_prog_row, row_major());
            multiply(*pi_view, dyn_prog_prev_row, dyn_prog_row);
            // emission
            // print(dyn_prog_row);
            // array1d<float, device_memory> expanded_lex(p_indexer -> get_state_size(), 0.0f);
            // cout << '5' << endl;
            array2d<float, device_memory>::column_view lex_column = lex_view -> column(token);
            // print(lex_column);
            // cout << '6' << endl;
            multiply(* lexMultiplier, lex_column, * expanded_lex);
            // print(expanded_lex);
            // cout << '7' << endl;
            blas::xmy(*expanded_lex, dyn_prog_row, dyn_prog_row);
            // cout << '8' << endl;
            // copy(temp_dyn_prog_row, dyn_prog_row);
        }
        normalizer = thrust::reduce(thrust::device, dyn_prog_row.begin(), dyn_prog_row.end());
        // print(dyn_prog_row);
        blas::scal(dyn_prog_row, 1.0f/normalizer);
        // print( dyn_prog_row);
        sentence_log_prob += log10f(normalizer);
        // cout << normalizer << sentence_log_prob << endl;
        i++;
        // if (i>1){
        //     throw;
        // }
        // skipping some error handling stuff
    }
    // auto t2 = Clock::now();
    // cout << "fpass: " << (float)std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count() * nano_to_sec << " s" << endl;
    return sentence_log_prob;
}

std::vector<State> HmmSampler::reverse_sample(std::vector<int> sent, int sent_index){
    // auto t2 = Clock::now();
    std::vector<State> sample_seq;
    std::vector<int> sample_t_seq;
    int last_index, sample_t, sample_depth; // , t, ind; totalK, depth,
    // int prev_depth, next_f_depth, next_awa_depth;
    //float sample_log_prob;//trans_prob, 
    // double t0, t1;
//    array1d<float, device_memory> trans_slice;
    State sample_state;
    //sample_log_prob = 0.0f;
    
    last_index = sent.size() - 1;
    // doubly normalized??
    // self.dyn_prog[last_index,:] /= self.dyn_prog[last_index,:].sum()
    sample_t = -1;
    sample_depth = -1;
    // cout << "x1" << endl;
    while (sample_t < 0 || (sample_depth > 0)) {
        // cout << "x2" << endl;
        array2d<float, device_memory>::row_view dyn_prog_temp_row_view = dyn_prog->row(last_index);
        sample_t = get_sample(dyn_prog_temp_row_view);
        // sample_t = 0;
        // cout << sample_t << endl;
        sample_state = p_indexer -> extractState(sample_t);
        // cout << sample_state.f << " " << sample_state.j << " " << sample_state.a[0] << " " << sample_state.a[1] << " " << sample_state.b[0] << " " << sample_state.b[1] << " " << sample_state.g << endl;
        sample_depth = sample_state.max_awa_depth();
        // cout << sample_depth << endl;
    }
    // auto t3 = Clock::now();
    // cout << "x3" << endl;
    sample_seq.push_back(sample_state);
    sample_t_seq.push_back(sample_t);
    // skip some error handling
    
    for (int t = sent.size() - 2; t > -1; t --){
        // cout << 't' << t << endl;
        // auto t11 = Clock::now();
        std::tie(sample_state, sample_t) = _reverse_sample_inner(sample_t, t);
        // cout << "Sample t is " << sample_t << endl;
        sample_seq.push_back(sample_state);
        sample_t_seq.push_back(sample_t);
        // auto t12 = Clock::now();
        // cout << "backpass2inside: " << (float)std::chrono::duration_cast<std::chrono::nanoseconds>(t12 - t11).count() * nano_to_sec << " s" << endl;

    }
    // auto t4 = Clock::now();
    std::reverse(sample_seq.begin(), sample_seq.end());
    // cout << '3' << endl;
    // cout << sample_seq.size() << endl;
    // auto t5 = Clock::now();
    // cout << "backpass1: " << (float)std::chrono::duration_cast<std::chrono::nanoseconds>(t3 - t2).count() * nano_to_sec << " s" << endl;
    // cout << "backpass2: " << (float)std::chrono::duration_cast<std::chrono::nanoseconds>(t4 - t3).count() * nano_to_sec << " s" << endl;
    // cout << "backpass: " << (float)std::chrono::duration_cast<std::chrono::nanoseconds>(t5 - t2).count() * nano_to_sec << " s" << endl;
    //for (int k : sample_t_seq){
    //    cout << sent_index << " : " << k  << endl;
    //}
    return sample_seq;
}


std::tuple<State, int> HmmSampler::_reverse_sample_inner(int& sample_t, int& t){
    //int ind;
    float normalizer;
    // auto t11 = Clock::now();
    get_row(pi->get_view(), sample_t, *trans_slice);
    // cout << "trans_slice" << endl;
    // print(*trans_slice); 
    // auto t12 = Clock::now();
    array2d<float, device_memory>::row_view dyn_prog_row = dyn_prog->row(t);
    // cout << "Dyn_prog_row" << endl;
    // print(dyn_prog_row);
    float trans_slice_sum = thrust::reduce(thrust::device, (*trans_slice).begin(), (*trans_slice).end());
    if (trans_slice_sum != 0.0){
        cusp::blas::xmy(*trans_slice, dyn_prog_row, dyn_prog_row);
    }
    // auto t13 = Clock::now();
    normalizer = thrust::reduce(thrust::device, dyn_prog_row.begin(), dyn_prog_row.end());
    // cout << "normalizer" << normalizer <<endl;
    blas::scal(dyn_prog_row, 1.0f/normalizer);
    // thrust::transform(dyn_prog_row.begin(), dyn_prog_row.end(), dyn_prog_row.begin(), multiplies_value<float>(1 / normalizer));
    // auto t14 = Clock::now();
    sample_t = get_sample(dyn_prog_row);
    //if (sample_t == 0){
    //    print(dyn_prog_row);
    //}
    // auto t15 = Clock::now();
    State sample_state = p_indexer -> extractState(sample_t);
    // cout << "backpass1reverseinner: " << (float)std::chrono::duration_cast<std::chrono::nanoseconds>(t12 - t11).count() * nano_to_sec << " s" << endl;
    // cout << "backpass1reverseinner: " << (float)std::chrono::duration_cast<std::chrono::nanoseconds>(t13 - t12).count() * nano_to_sec << " s" << endl;
    // cout << "backpass1reverseinner: " << (float)std::chrono::duration_cast<std::chrono::nanoseconds>(t14 - t13).count() * nano_to_sec << " s" << endl;
    // cout << "backpass1reverseinner: " << (float)std::chrono::duration_cast<std::chrono::nanoseconds>(t15 - t14).count() * nano_to_sec << " s" << endl;

    return std::make_tuple(sample_state, sample_t);
}

std::tuple<std::vector<State>, float> HmmSampler::sample(std::vector<int> sent, int sent_index) {
    
    float log_probs = forward_pass(sent, sent_index);
    
    std::vector<State> states = reverse_sample(sent, sent_index);
    
    
    return std::make_tuple(states, log_probs);
}

HmmSampler::HmmSampler() : seed(std::time(0)){
    mt.seed(seed);
}

HmmSampler::HmmSampler(int seed) : seed(seed){
    if (seed == 0){
        mt.seed(std::time(0));
    } else{
        mt.seed(seed);
    }
}
HmmSampler::~HmmSampler(){
    //delete p_model;
    delete p_indexer;
    //delete lexMatrix;
    delete dyn_prog;
    delete lexMultiplier;
    //delete pi;
    delete trans_slice;
    //delete expanded_lex;
    //delete sum_dict;
}
