#include "hip/hip_runtime.h"

#include "HmmSampler.h"
#include <stdio.h>
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>
#include <thrust/scatter.h>
#include <tuple>
#include <ctime>
#include <utility>
#include <cmath>
#include <algorithm>
#include "cusp/array1d.h"
#include "cusp/array2d.h"
#include "cusp/csr_matrix.h"
#include "cusp/blas/blas.h"
#include "cusp/multiply.h"
#include "cusp/elementwise.h"
#include "cusp/functional.h"
#include "cusp/print.h"
#include "myarrays.cu"
#include "Indexer.cu"
#include "State.cu"
#include <chrono> // for testing
#include <random>
#include <limits>
#include <iostream>
#include <iomanip>
using namespace cusp;
using namespace std;
typedef std::chrono::high_resolution_clock Clock;
float nano_to_sec = 1.0e-9f;
__device__ int G_SIZE;
typedef std::numeric_limits<float> float_limit;
typedef cusp::array1d<float,cusp::device_memory> cusparray;
typedef cusparray::view ArrayView;

Model::Model(int pi_num_rows, int pi_num_cols, float* pi_vals, int pi_vals_size, int* pi_row_offsets,
int pi_row_offsets_size, int* pi_col_indices, int pi_col_indices_size, float* lex_vals, int lex_vals_size,
int lex_num_rows, int lex_num_cols, int a_max, int b_max, int g_max, int depth, float* pos_vals, int pos_vals_size,
int EOS_index):
pi_num_rows(pi_num_rows), pi_num_cols(pi_num_cols), pi_vals(pi_vals),
pi_vals_size(pi_vals_size), pi_row_offsets(pi_row_offsets), pi_row_offsets_size(pi_row_offsets_size),
pi_col_indices(pi_col_indices), pi_col_indices_size(pi_col_indices_size), lex_vals(lex_vals),
lex_vals_size(lex_vals_size), lex_num_rows(lex_num_rows), lex_num_cols(lex_num_cols), a_max(a_max),
b_max(b_max), g_max(g_max), depth(depth), pos_vals(pos_vals), pos_vals_size(pos_vals_size), EOS_index(EOS_index){
    pi = new SparseView(pi_num_rows, pi_num_cols, pi_vals_size, pi_row_offsets, pi_col_indices, pi_vals, pi_row_offsets_size, pi_col_indices_size);
    lex = new DenseView(lex_num_rows, lex_num_cols, lex_vals_size, lex_vals);
    pos = new Array(pos_vals, pos_vals_size);
}

Model::~Model(){
    delete lex;
    delete pi;
    delete pos;
}

int Model::get_depth(){
    return depth;
}

// taken away the new tensor
template <class AView>
int HmmSampler::get_sample(AView &v){
    float dart;
//    cusp::print(v);
//     cout << "get_sample()" << endl;
    // array1d<float, device_memory> sum_dict(v.size()); // building a new array, maybe not needed
    // this is the equivalent of np.cumsum() or partial_sum in the stl:
    thrust::inclusive_scan(thrust::device, v.begin(), v.end(), sum_dict->begin());
    // cout << "v[0] = " << v[0] << "v[-1] = " << v[v.size()-1] << " with length " << v.size() << endl;
//     cout << "sum_dict[0] = " << (*sum_dict)[0] << "sum_dict[-1] = " << (*sum_dict)[sum_dict->size()-1] << " with length: " << sum_dict->size() <<  endl;
    //cusp::print(*sum_dict);
//     cout << "sum done" << endl;
    // dart =  static_cast <float> (rand()) / static_cast <float> (RAND_MAX);// / RAND_MAX;
    int dart_target;
    int condition = 1;
    while (condition) {
        dart = dist(mt);
        if (dart != 0.0f and dart != 1.0f) {
//            cout << "dart: "<< scientific << dart << endl;
           dart_target = thrust::upper_bound(thrust::device, sum_dict->begin(), sum_dict->end(), dart) - sum_dict->begin();
           if (dart_target == 0) {
           print(*sum_dict);
           }
//           cout << "dart target (summed): " << dart_target << " " << scientific << (*sum_dict)[dart_target - 1] << endl; //<< " " << scientific << (*sum_dict)[dart_target] << " " << scientific  << (*sum_dict)[dart_target+ 1] << endl;
//           cout << "dart target (summed): " << dart_target << " " << scientific << v[dart_target - 1] << " " << scientific << v[dart_target] << " " << scientific  << v[dart_target+ 1] << endl;
//            cout << "dart target (summed): " << dart_target << " with v size=" << v.size() << endl;
//            float minus_one = (*sum_dict)[dart_target - 1];
//            printf( "%A" , minus_one);
//            cout  << " ";
//            float on_target = (*sum_dict)[dart_target];
//            printf("%A", on_target);
//            cout << " ";
//            float plus_one = (*sum_dict)[dart_target+ 1];
//            printf("%A\n", plus_one);
           if (dart_target != v.size() && v[dart_target] != 0.0f ){
                condition = 0;
//                cout << "out of loop" << endl;
           }
        }
     }
//      cout << "get_sample() done." << endl;
//      cout << "dart_target: " << dart_target << endl;
     return dart_target;
}

class cuda_exp : public thrust::unary_function<float, float>{
public:
    __device__
   float operator()(float x){
       return pow(10,x);
   }
};
class calc_sparse_column : public thrust::unary_function<int, int>{
public:
   __device__
   int operator()(int x){
       return x % G_SIZE;
   }
};

class g_integer_division : public thrust::unary_function<int, int>{
public:
    __device__
   int operator()(int x){
       return x / G_SIZE;
   }
};

template <class AView>
void exp_array(AView & v){
   cuda_exp x;
   thrust::transform(v.begin(), v.end(), v.begin(), x);
}

// this gives a TRANSPOSED version of a tile matrix for computation convenience
Sparse * tile(int g_len, int state_size){

//     cout << "5.1" << endl;
//    int copy_times = state_size / g_len;
    // cout << "5.2" << endl;
    int nnz = state_size;
//     cout << "5.3" << endl;
    // cout << g_len << " " << state_size << " " << nnz << endl;
    Sparse * temp_mat = new Sparse(state_size, g_len, nnz); //TRANSPOSED
    blas::fill(temp_mat->values, 1.0f);
//     cout << "5.4" << endl;
    calc_sparse_column temp_func;
    thrust::tabulate(thrust::device, temp_mat->column_indices.begin(), temp_mat->column_indices.end(), temp_func);
    thrust::sequence(thrust::device, temp_mat->row_offsets.begin(), temp_mat->row_offsets.end());
    // for (int i = 0; i <= state_size; i++){
    //     if (i < state_size){
    //         temp_mat->row_offsets[i] = i;
    //     } else {
    //         temp_mat->row_offsets[i] = nnz;
    //     }
    // }
//     cout << "5.5" << endl;

    return temp_mat;
}

// this gives a TRANSPOSED version of matrix repetition for computation convenience
Sparse * expand(int g_len, int state_size){

//     cout << "expand5.1" << endl;
    int col_size = state_size / g_len;
//     cout << "expand5.2" << endl;
    int nnz = state_size;
//     cout << "expand5.3" << endl;
    // cout << g_len << " " << state_size << " " << nnz << endl;
    Sparse * temp_mat = new Sparse(state_size, col_size, nnz); //TRANSPOSED
    blas::fill(temp_mat->values, 1.0f);
//     cout << "expand5.4" << endl;
    g_integer_division temp_func;
    thrust::tabulate(thrust::device, temp_mat->column_indices.begin(), temp_mat->column_indices.end(), temp_func);
    thrust::sequence(thrust::device, temp_mat->row_offsets.begin(), temp_mat->row_offsets.end());
//     cout << "expand5.5" << endl;
    return temp_mat;
}

void get_row(csr_matrix_view<IndexArrayView, IndexArrayView, ValueArrayView>* s, int i, Array& result,
Array* pos_full_array, int g_max, int b_max){
    blas::fill(result, 0.0f);
    int pi_row_index = i / g_max;
    int b_index = pi_row_index % b_max;
    int g_index = i % g_max;
//    cout <<"get row in"<< endl;
//    cout << pi_row_index << "pi row index" << endl;
    if (s->row_offsets[pi_row_index] - s->row_offsets[pi_row_index+1] != 0){
        int num_of_entry = s->row_offsets[pi_row_index+1] - s->row_offsets[pi_row_index];
        int start_column_index = s->row_offsets[pi_row_index];
        IndexArrayView column_indices = s -> column_indices.subarray(start_column_index, num_of_entry);
        ValueArrayView values = s -> values.subarray(start_column_index, num_of_entry);
        thrust::scatter(thrust::device, values.begin(), values.end(), column_indices.begin(), result.begin());
//        cusp::print(result);
//        cout << "g is " << g_index << "and the prob is " <<(*pos_matrix)[pos_matrix_dim] << endl;
//        blas::scal(result, (*pos_full_array)[i]);
    }
//    cout << "get row out" << endl;
}

int get_max_len(std::vector<std::vector<int> > sents){
//    cout <<"get_max_len in"<< endl;
    int max_len = 0;
    for(std::vector<int> sent : sents){
        if(sent.size() > max_len){
            max_len = sent.size();
        }
    }
//    cout << "get_max_len out" << endl;

    return max_len;
}

Dense* get_sentence_array(std::vector<std::vector<int> > sents, int max_len){
    Dense* array = new Dense( sents.size(), max_len, 0 );
//    cout <<"get_sentence_array in"<< endl;
    for(int i = 0; i < sents.size(); i++){
      std::vector<int> sent = sents[i];
      for(int token_ind = 0; token_ind < sent.size(); token_ind++){
        array -> operator()(i, token_ind) = sent[token_ind];
      }
    }
//    cout <<"get_sentence_array out"<< endl;
    return array;

}

void HmmSampler::set_models(Model * models){

//     cout << "set_models 1" << endl;
    p_model = models;
//     cout << "set_models 2" << endl;
    if (p_indexer != NULL){
        delete p_indexer;
	p_indexer = NULL;
    }
    p_indexer = new Indexer(models);
//     cout << "set_models 3" << endl;
    int g_len = p_model-> g_max;
    int state_size = p_indexer -> get_state_size();
    hipMemset(&G_SIZE,0,sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(G_SIZE), &g_len, sizeof(int), 0, hipMemcpyHostToDevice);
//     cout << "set_models 4" << endl;
    //if (lexMatrix != NULL){
    //    delete lexMatrix;
    //    lexMatrix = NULL;
    //}

    lexMatrix = p_model -> lex;
    // print(*(lexMatrix->get_view()));
    // exp_array(lexMatrix->get_view()->values); // exp the lex dist // the gpu models are not logged, should not need this
//     cout << "set_models 5" << endl;
    pos_matrix = p_model -> pos;
//    cusp::print(*pos_matrix);
    if (lexMultiplier != NULL){
        delete lexMultiplier;
        lexMultiplier = NULL;
    }
    lexMultiplier = tile(g_len, p_indexer->get_state_size());
//    cout << "set_models 6" << endl;
    expand_mat = expand(g_len, p_indexer->get_state_size());
//    cout << "set_models 7" << endl;
    // print(*lexMultiplier);
    pi = p_model -> pi;
    // print( *(pi->get_view()) );
    if (trans_slice != NULL){
        delete trans_slice;
        trans_slice = NULL;
    }
    trans_slice = new Array(p_indexer->get_state_size(), 0.0f);
//    cout << "set_models 8" << endl;
    expanded_lex = trans_slice;
    sum_dict = trans_slice;
    int b_len = p_model -> b_max;
//    cout << "pos full array" << endl;
    int depth = p_model -> get_depth();
    pos_full_array = make_pos_full_array(pos_matrix, g_len, b_len, depth, state_size);
//    cout << "set_models 9" << endl;
//    print(*pos_full_array);
    // cout.precision(float_limit::max_digits10);
}

Array* HmmSampler::make_pos_full_array(Array* pos_matrix ,int g_max, int b_max, int depth, int state_size){
//    cout << "make_pos_full_array in" << endl;
    int pos_matrix_size = pow(b_max, depth) * g_max * 2;
    int copy_times = state_size / pos_matrix_size;
    Array* temp_array = new Array(state_size, 0.0f);
    for (int i = 0; i < copy_times; i ++){
//        cout << "calc full pos " << i  << " from " << i*pos_matrix_size << " to " << (i+1)*pos_matrix_size <<
//        " total " << copy_times << endl;
        ArrayView one_section_of_array(temp_array->subarray(i*pos_matrix_size, pos_matrix_size));
        copy(*pos_matrix, one_section_of_array);
    }
//    cout << "make_pos_full_array out" << endl;
//    cout << " THE VALUE " << (*temp_array)[26426] << endl;
    return temp_array;
}

void HmmSampler::initialize_dynprog(int batch_size, int max_len){
    try{
//    cout << "initialize_dynprog in" << endl;
    sampler_batch_size = batch_size;
    max_sent_len = max_len;
    dyn_prog = new Dense*[max_len];
    for(int i = 0; i < max_len; i++){
        dyn_prog[i] = new Dense(p_indexer->get_state_size(), batch_size, 0.0f);
    }
//    cout << "initialize_dynprog 2" << endl;
    start_state = new Dense(p_indexer->get_state_size(), batch_size, 0.0f);
    for(int i = 0; i < batch_size; i++){
        start_state->operator()(0, i) = 1;
    }
    int a_max, b_max, g_max;
//    cout << "initialize_dynprog 3" << endl;
    std::tie(a_max, b_max, g_max) = p_indexer -> getVariableMaxes();
    int state_size_no_g = p_indexer->get_state_size() / g_max;
    dyn_prog_part = new Dense(state_size_no_g, batch_size, 0.0f);
//    cout << "initialize_dynprog out" << endl;
    } catch (...) {
        cout << "init dynprog error!" << endl;
        throw;
    }
}

void HmmSampler::g_factored_multiply(Dense* prev_dyn_prog_slice, Dense* this_dyn_prog_slice){
//    cout << "factor multiply 1" << endl;
    csr_matrix_view<IndexArrayView,IndexArrayView,ValueArrayView>* curtailed_transition = pi -> get_view();
    multiply(*curtailed_transition, *prev_dyn_prog_slice, *dyn_prog_part);
//    cout << "dyn prog part" << endl;
//    print(*dyn_prog_part);
//    cout << "factor multiply 2" << endl;
    int a_max, b_max, g_max;
    std::tie(a_max, b_max, g_max) = p_indexer -> getVariableMaxes();
    int state_size_no_g = p_indexer->get_state_size() / g_max;
    int state_size = p_indexer->get_state_size();
    multiply(*expand_mat, *dyn_prog_part, *this_dyn_prog_slice);
//    cout << "factor multiply 3" << endl;
//    cout << "this dyn prog slice" << endl;
//    print(*this_dyn_prog_slice);
    for(int i = 0; i < sampler_batch_size; i++ ){ // this is not efficient. maybe there is a better way
//        cout << "insde g_multipy i " << i << endl;
        array2d<float, device_memory>::column_view this_dyn_prog_slice_column = this_dyn_prog_slice->column(i);
//        blas::xmy(this_dyn_prog_slice_column, *pos_matrix, this_dyn_prog_slice_column);
//        print(this_dyn_prog_slice_column);
//        print(*pos_full_array);
        blas::xmy(this_dyn_prog_slice_column, *pos_full_array ,this_dyn_prog_slice_column);
//        cout << "this dyn prog slice column" << endl;
//        print(this_dyn_prog_slice_column);

    }
//    cout << "factor multiply 4" << endl;
}

std::vector<float> HmmSampler::forward_pass(std::vector<std::vector<int> > sents, int sent_index){
    // auto t1 = Clock::now();
//     cout << "Forward in " << endl;
    float normalizer;
    int a_max, b_max, g_max; // index, token, g_len;
    std::tie(a_max, b_max, g_max) = p_indexer -> getVariableMaxes();
    std::vector<int> sent = sents[0];
    std::vector<float> log_probs;
    int batch_size = sents.size();
    int batch_max_len = get_max_len(sents);
    // np_sents is |batches| x max_len
    Dense* np_sents = get_sentence_array(sents, batch_max_len);

    array2d_view<ValueArrayView, row_major>* lex_view = lexMatrix -> get_view();
//    cout << "Forward in 2 " << endl;
    // initialize likelihood vector:
    for(int sent_ind = 0; sent_ind < sents.size(); sent_ind++){
        log_probs.push_back(0);
    }


    for(int ind = 0; ind < batch_max_len; ind++){
//        cout << "Processing token index " << ind << " for " << batch_size << " sentences." << endl;
        Dense *cur_mat = dyn_prog[ind];
        Dense *prev_mat;

        if(ind == 0){
            prev_mat = start_state;
        }else{
            // Grab the ind-1th row of dyn_prog and multiply it by the transition matrix and put it in
            // the ind^th row.
            prev_mat = dyn_prog[ind-1];
        }

        // prev_mat is |states| x |batches| at time ind-1
        // pi_view is |states| x |states| transition matrix with time t on rows and t-1 on columns (i.e. transposed)
        // so after this multiply cur_mat is |states| x |batches| incorporating transition probabilities
        // but not evidence
//        cout << "Performing transition multiplication" << endl;
        g_factored_multiply(prev_mat, cur_mat);
//        print(cur_mat[0]);
//        cout << "Done with transition" << endl;
//        cout << "performing observation multiplications" << endl;

        auto trans_done = Clock::now();

        // for now incorporate the evidence sentence-by-sentence:
        for(int sent_ind = 0; sent_ind < sents.size(); sent_ind++){
            // not every sentence in the batch will need the full batch size
            if(sents[sent_ind].size() <= ind){
                continue;
            }
//            cout << "Processing sentence index " << sent_ind << endl;
            int token = sents[sent_ind][ind];

            // lex_column is |g| x 1
            array2d<float, device_memory>::column_view lex_column = lex_view -> column(token);
//            if (sents[sent_ind].size() > 3){
//                cout << "lex"<< endl;
//                print(lex_column);
//            }
//             cout << "6" << endl;
            // lexMultiplier is state_size x |g|, expanded_lex is state_size x 1
//             cout << "Multiplying lex multiplier by lex column" << endl;
//            cout << "lex column" << endl;
//            print(lex_column);
            multiply(* lexMultiplier, lex_column, * expanded_lex);
//            if (sents[sent_ind].size() > 3){
//                cout << "expanded lex" << endl;
//                print(*expanded_lex);
//            }

//             cout << "lex finished" << endl;
            // dyn_prog_row is 1 x state_size
            // dyn_prog_column is state_size x 1
            array2d<float, device_memory>::column_view dyn_prog_col = cur_mat->column(sent_ind);
//             cout << "Multiplying expanded_lex by dyn prog row" << endl;
//            vector<int> v = {8, 19406, 24314, 26426};
//            if (sents[sent_ind].size() > 3 && ind < 4){
//                cout << "column view" << endl;
//                cout << "ind" << ind << " " << dyn_prog_col[v[ind]] << endl;
//                cout << "lex" << ind << " " << (*expanded_lex)[v[ind]] << endl;
//                cout << "pos" << ind << " " << (*pos_full_array)[v[ind]] << endl;
//            }
            blas::xmy(*expanded_lex, dyn_prog_col, dyn_prog_col);
//             cout << "Computing normalizer" << endl;
            normalizer = thrust::reduce(thrust::device, dyn_prog_col.begin(), dyn_prog_col.end());
//             cout << "Normalizing over col with result: " << normalizer << endl;
//            cout << "Scaling by normalizer" << endl;
            blas::scal(dyn_prog_col, 1.0f/normalizer);
//            cout << "normed column" << endl;
//            print(dyn_prog_col);
//                cout << "Adding logged normalizer to sentence logprobs" << endl;
            log_probs[sent_ind] += log10f(normalizer);
//            cout << " ind "<< ind << " sent_ind " << sent_ind << " token " << token << " log prob " << log10f(normalizer) << " " << normalizer << endl;
            if (sents[sent_ind].size() - 1 == ind){
                int EOS = p_indexer -> get_EOS_full();
//                cout << EOS << endl;
                array2d<float, device_memory>::column_view final_dyn_col = cur_mat->column(sent_ind);
//                cout << cusp::blas::asum(final_dyn_col) << endl;
                get_row(pi->get_view(), EOS, *trans_slice, pos_full_array, g_max, b_max);
//                cout << cusp::blas::asum(*trans_slice) << endl;
                float final_normalizer = cusp::blas::dot(*trans_slice, final_dyn_col);
                log_probs[sent_ind] += log10f(final_normalizer);
//                cout << " ind "<< ind << " sent_ind " << sent_ind << "end log prob " << log10f(final_normalizer) << endl;
            }
        }

        auto norm_done = Clock::now();
    }
//    for (int i =0; i < sents[0].size(); i++){
//    print(*dyn_prog[i]);
//    }
//    cout << "end of dyn prog" << endl;
//    cout << "Finished forward pass (cuda) and returning vector with " << log_probs.size() << " elements." << endl;
    return log_probs;
}

std::vector<std::vector<State> > HmmSampler::reverse_sample(std::vector<std::vector<int>> sents, int sent_index){
//     cout << "Reverse sampling batch with starting sent index of " << sent_index << endl;
    // auto t2 = Clock::now();
    std::vector<std::vector<State>> sample_seqs;
    std::vector<State> sample_seq;
    std::vector<int> sample_t_seq;
    int last_index, sample_t, sample_depth; // , t, ind; totalK, depth,
    int batch_size = sents.size();
    int batch_max_len = get_max_len(sents);
    // int prev_depth, next_f_depth, next_awa_depth;
    //float sample_log_prob;//trans_prob,
    // double t0, t1;
//    array1d<float, device_memory> trans_slice;
    State sample_state;
    //sample_log_prob = 0.0f;
//    std::vector<int> fake_ts = {25,218,21,213,25,230,42,22,38,229,233,41,25,154,2};
//    reverse(fake_ts.begin(), fake_ts.end());
//    for(std::vector<int> sent : sents){
    for(int sent_ind = 0; sent_ind < batch_size; sent_ind++){
        sample_seq = std::vector<State>();
//        cout << "Processing sentence " << sent_ind << " of the batch" << endl;
        std::vector<int> sent = sents[sent_ind];
//        for(int token_ind = 0; token_ind < sent.size(); token_ind++){
//          cout << sent[token_ind] << " ";
//        }
//        cout << endl;

        // Start with EOS
//        if (sent.size() == 1) {
//            sample_t = p_indexer->get_EOS_1wrd_full();
//        } else {
            sample_t = p_indexer->get_EOS_full();
//        }

        for (int t = sent.size() - 1; t > -1; t --){
//            cout << "t" << t << " prev sample t is " << sample_t <<endl;
            // auto t11 = Clock::now();
            std::tie(sample_state, sample_t) = _reverse_sample_inner(sample_t, t, sent_ind);
//             cout << "Sample t is " << sample_t << endl;
//             cout << sample_state.f << " " << sample_state.j << " " << sample_state.a[0] << " " << sample_state.a[1] << " " << sample_state.b[0] << " " << sample_state.b[1] << " " << sample_state.g << endl;
            if(!sample_state.depth_check()){
              cout << "Depth error in state assigned at index" << t << endl;
            }
            sample_seq.push_back(sample_state);
            sample_t_seq.push_back(sample_t);
            // auto t12 = Clock::now();
//             cout << "backpass2inside: " << (float)std::chrono::duration_cast<std::chrono::nanoseconds>(t12 - t11).count() * nano_to_sec << " s" << endl;

        }
        // auto t4 = Clock::now();
        std::reverse(sample_seq.begin(), sample_seq.end());
//        cout << "finished backward sampling of a sentence" << endl;
        //cout << sample_seq->size() << endl;
        // auto t5 = Clock::now();
        // cout << "backpass1: " << (float)std::chrono::duration_cast<std::chrono::nanoseconds>(t3 - t2).count() * nano_to_sec << " s" << endl;
        // cout << "backpass2: " << (float)std::chrono::duration_cast<std::chrono::nanoseconds>(t4 - t3).count() * nano_to_sec << " s" << endl;
        // cout << "backpass: " << (float)std::chrono::duration_cast<std::chrono::nanoseconds>(t5 - t2).count() * nano_to_sec << " s" << endl;
        //for (int k : sample_t_seq){
        //    cout << sent_index << " : " << k  << endl;
        //}
        sample_seqs.push_back(sample_seq);
    }
//    for(auto i : sample_t_seq){
//        cout << i << endl;
//    }
//    for (int i =0; i < sents[0].size(); i++){
//    print(*dyn_prog[i]);
//    }
//    cout << "end of dyn prog" << endl;
//    cout << "Done with reverse()" << endl;
    return sample_seqs;
}


std::tuple<State, int> HmmSampler::_reverse_sample_inner(int& sample_t, int& t, int sent_ind){
    //int ind;
    float normalizer;
    // auto t11 = Clock::now();
//    cout << "before get row" << endl;
//    int prev_sample_t = sample_t;
    int g_max =  p_model -> g_max;
    int b_max = p_model -> b_max;
    get_row(pi->get_view(), sample_t, *trans_slice, pos_full_array, g_max, b_max);
    // print(*trans_slice);
    // auto t12 = Clock::now();
    array2d<float, device_memory>::column_view dyn_prog_col = dyn_prog[t]->column(sent_ind);
    float dyn_prog_col_sum = thrust::reduce(thrust::device, dyn_prog_col.begin(), dyn_prog_col.end());
//     cout << "Dyn_prog_row " << dyn_prog_col_sum << endl;
    // print(dyn_prog_row);
    float trans_slice_sum = thrust::reduce(thrust::device, (*trans_slice).begin(), (*trans_slice).end());
//    cout << "trans_slice " << trans_slice_sum << endl;
    //if (trans_slice_sum != 0.0f){
        cusp::blas::xmy(*trans_slice, dyn_prog_col, dyn_prog_col);
    //}
    // auto t13 = Clock::now();
//    cusp::array1d<float, host_memory> un_normalized_sums(dyn_prog_col);
    normalizer = thrust::reduce(thrust::device, dyn_prog_col.begin(), dyn_prog_col.end());
//     cout << "normalizer " << normalizer <<endl;
    blas::scal(dyn_prog_col, 1.0f/normalizer);
    // thrust::transform(dyn_prog_row.begin(), dyn_prog_row.end(), dyn_prog_row.begin(), multiplies_value<float>(1 / normalizer));
    // auto t14 = Clock::now();
    sample_t = get_sample(dyn_prog_col);
//    cout << "sample_t "<< sample_t << endl;
//    sample_t = fake_ts.back();
//    fake_ts.pop_back();
    //if (sample_t == 0){
    //    print(dyn_prog_row);
    //}
    // auto t15 = Clock::now();
    State sample_state = p_indexer -> extractState(sample_t);
//    cout << "done with reverse sampler inner" << endl;
    // if ((sample_state.a[1] == 0 && sample_state.b[1] != 0)|| (sample_state.a[1] != 0 && sample_state.b[1] == 0) ){
//    get_row(pi->get_view(), prev_sample_t, *trans_slice); // why is this used?
        // cout << "Dyn Prog(normalized): " << scientific << dyn_prog_row[sample_t -1] << " " << scientific << dyn_prog_row[sample_t] <<" "<< scientific <<dyn_prog_row[sample_t + 1] <<endl;
        // cout << "Trans Slice (Pi Row): " <<" " << scientific << (*trans_slice)[sample_t-1]<< " "<< scientific << (*trans_slice)[sample_t] <<" " << scientific << (*trans_slice)[sample_t+1]<< endl;
        // cout << "Prefix Sum(unnormalized, before sum): " << scientific << un_normalized_sums[sample_t-1] << " " << scientific << un_normalized_sums[sample_t] << " " << scientific << un_normalized_sums[sample_t + 1] << endl;
        // cout << "Prefix Sum(unnormalozed, before sum): ";
        // float temp_1 = un_normalized_sums[sample_t-1];
        // printf( "%A", temp_1);
        // cout << " ";
        // float temp_2 = un_normalized_sums[sample_t];
        // printf("%A", temp_2);
        // cout << " ";
        // float temp_3 = un_normalized_sums[sample_t+1];
        // printf("%A\n", temp_3);
        // cout << "sample_t: " << sample_t << endl;

    // }
    // cout << "backpass1reverseinner: " << (float)std::chrono::duration_cast<std::chrono::nanoseconds>(t12 - t11).count() * nano_to_sec << " s" << endl;
    // cout << "backpass1reverseinner: " << (float)std::chrono::duration_cast<std::chrono::nanoseconds>(t13 - t12).count() * nano_to_sec << " s" << endl;
    // cout << "backpass1reverseinner: " << (float)std::chrono::duration_cast<std::chrono::nanoseconds>(t14 - t13).count() * nano_to_sec << " s" << endl;
    // cout << "backpass1reverseinner: " << (float)std::chrono::duration_cast<std::chrono::nanoseconds>(t15 - t14).count() * nano_to_sec << " s" << endl;

    return std::make_tuple(sample_state, sample_t);
}

std::tuple<std::vector<std::vector<State> >, std::vector<float>> HmmSampler::sample(std::vector<std::vector<int>> sents, int sent_index) {
    std::vector<float> log_probs;
    std::vector<std::vector<State> > states;

    try{
        log_probs = forward_pass(sents, sent_index);
    }catch(thrust::system_error &e){
        cerr << "Error in forward pass: " << e.what() << endl;
        throw e;
    }
    try{
        states = reverse_sample(sents, sent_index);
    }catch(thrust::system_error &e){
        cerr << "Error in reverse sample: " << e.what() << endl;
        throw e;
    }


    return std::make_tuple(states, log_probs);
}

HmmSampler::HmmSampler() : seed(std::time(0)){
    mt.seed(seed);
}

HmmSampler::HmmSampler(int seed) : seed(seed){
    if (seed == 0){
        mt.seed(std::time(0));
    } else{
        mt.seed(seed);
    }
}
HmmSampler::~HmmSampler(){
    if(dyn_prog != NULL){
        for(int i = 0; i < max_sent_len; i++){
            delete dyn_prog[i];
        }
        delete[] dyn_prog;
    }
    delete start_state;
    //delete[] dyn_prog;
    //delete p_model;
    delete p_indexer;
    //delete lexMatrix;
    //delete dyn_prog;
    delete lexMultiplier;
    //delete pi;
    delete trans_slice;
    //delete expanded_lex;
    //delete sum_dict;
    delete expand_mat;
    delete dyn_prog_part;
    delete pos_full_array;

}
